
#include <hip/hip_runtime.h>
#ifdef GOOGLE_CUDA
#define EIGEN_USE_GPU
#include "linear_op.h"
#include "cuda_utils.h"

using namespace tfinterp;

using GPUDevice = Eigen::GpuDevice;

// Define the CUDA kernel.
template <typename T>
__global__ void LinearInterpCUDAKernel(int size, int M, const T* const x, const T* const y, int N, const T* const t, T* v, int* inds) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < size; i += stride) {
    int k = i / N;
    int off_m = k * M;
    inds[i] = interp::interp1d<T>(M, x + off_m, y, t[i], v + i);
  }
}

// Define the GPU implementation that launches the CUDA kernel.
template <typename T>
void LinearInterpCUDAFunctor<T>::operator()(
    const GPUDevice& d, int size, int M, const T* const x, const T* const y, int N, const T* const t, T* v, int* inds) {
  CudaLaunchConfig config = GetCudaLaunchConfig(size, d);
  int block_count = config.block_count;
  int thread_per_block = config.thread_per_block;
  LinearInterpCUDAKernel<T>
      <<<block_count, thread_per_block, 0, d.stream()>>>(size, M, x, y, N, t, v, inds);
}

template struct LinearInterpCUDAFunctor<float>;
template struct LinearInterpCUDAFunctor<double>;

#endif  // GOOGLE_CUDA
